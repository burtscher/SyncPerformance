#include "hip/hip_runtime.h"
/*
This file is part of SyncPerformance, a testing framework that measures the execution time of single synchronization primitives in CUDA and OpenMP.

BSD 3-Clause License

Copyright (c) 2024, Brandon Alexander Burtchell and Martin Burtscher
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

1. Redistributions of source code must retain the above copyright notice, this
   list of conditions and the following disclaimer.

2. Redistributions in binary form must reproduce the above copyright notice,
   this list of conditions and the following disclaimer in the documentation
   and/or other materials provided with the distribution.

3. Neither the name of the copyright holder nor the names of its
   contributors may be used to endorse or promote products derived from
   this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

URL: The latest version of this code is available at https://github.com/burtscher/SyncPerformance.

Publication: This work is described in detail in the following paper.
Brandon Alexander Burtchell and Martin Burtscher. "Characterizing CUDA and OpenMP Synchronization Primitives." Proceedings of the IEEE International Symposium on Workload Characterization. September 2024.
*/


#include <hip/hip_runtime.h>
#include <cstdio>
#include "cuda_macros.cuh"

int getSPcores(hipDeviceProp_t devProp);

int main() {
  hipSetDevice(0);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  if ((deviceProp.major == 9999) && (deviceProp.minor == 9999)) {
    fprintf(stderr, "ERROR: no CUDA capable device detected\n\n");
    exit(-1);
  }

  const int SMs = deviceProp.multiProcessorCount;
  const int mTpSM = deviceProp.maxThreadsPerMultiProcessor;
  const int blocks = SMs * (mTpSM / 1024);

  printf("GPU: %s\n", deviceProp.name);
  printf("  SMs: %d\n", SMs);
  printf("  mTpSM: %d\n", mTpSM);
  printf("  max blocks: %d\n", blocks);
  printf("  cores per SM: %d\n", getSPcores(deviceProp) / SMs);
  printf("  clock rate: %d\n", deviceProp.clockRate);  // NOTE: deprecated?
  printf("  regs per SM: %d\n", deviceProp.regsPerMultiprocessor);
  printf("  regs per block: %d\n", deviceProp.regsPerBlock);

  CheckCuda();

  return 0;
}

/**
 * https://stackoverflow.com/a/32531982
 */
int getSPcores(hipDeviceProp_t devProp) {
  int cores = 0;
  int mp = devProp.multiProcessorCount;
  switch (devProp.major) {
    case 2:  // Fermi
      if (devProp.minor == 1)
        cores = mp * 48;
      else
        cores = mp * 32;
      break;
    case 3:  // Kepler
      cores = mp * 192;
      break;
    case 5:  // Maxwell
      cores = mp * 128;
      break;
    case 6:  // Pascal
      if ((devProp.minor == 1) || (devProp.minor == 2))
        cores = mp * 128;
      else if (devProp.minor == 0)
        cores = mp * 64;
      else
        printf("Unknown device type\n");
      break;
    case 7:  // Volta and Turing
      if ((devProp.minor == 0) || (devProp.minor == 5))
        cores = mp * 64;
      else
        printf("Unknown device type\n");
      break;
    case 8:  // Ampere
      if (devProp.minor == 0)
        cores = mp * 64;
      else if (devProp.minor == 6)
        cores = mp * 128;
      else if (devProp.minor == 9)
        cores = mp * 128;  // ada lovelace
      else
        printf("Unknown device type\n");
      break;
    case 9:  // Hopper
      if (devProp.minor == 0)
        cores = mp * 128;
      else
        printf("Unknown device type\n");
      break;
    default:
      printf("Unknown device type\n");
      break;
  }
  return cores;
}